#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

// secp256k1 constants
__constant__ uint64_t SECP256K1_P[4] = {0xFFFFFFFFFFFFFFFFULL, 0xFFFFFFFFFFFFFFFFULL, 0xFFFFFFFFFFFFFFFFULL, 0xFFFFFFFFFFFFFC2FULL};
__constant__ uint64_t SECP256K1_N[4] = {0xFFFFFFFFFFFFFFFFULL, 0xFFFFFFFFFFFFFFFFULL, 0xFFFFFFFFFFFFFFFFULL, 0xFFFFFFFFFFFFFFFEULL, 0xBAAEDCE6AF48A03BULL, 0xBFD25E8CD0364141ULL};
__constant__ uint64_t SECP256K1_GX[4] = {0x79BE667EF9DCBBACULL, 0x55A06295CE870B07ULL, 0x029BFCDB2DCE28D9ULL, 0x59F2815B16F81798ULL};
__constant__ uint64_t SECP256K1_GY[4] = {0x483ADA7726A3C465ULL, 0x5DA4FBFC0E1108A8ULL, 0xFD17B448A6855419ULL, 0x9C47D08FFB10D4B8ULL};

// Utility functions for u256 arithmetic
__device__ void u256_add(uint64_t *res, const uint64_t *a, const uint64_t *b) {
    uint64_t carry = 0;
    for (int i = 0; i < 4; i++) {
        uint64_t temp = a[i] + carry;
        res[i] = temp + b[i];
        carry = (temp < a[i]) || (res[i] < temp);
    }
}

__device__ void u256_sub(uint64_t *res, const uint64_t *a, const uint64_t *b) {
    uint64_t borrow = 0;
    for (int i = 0; i < 4; i++) {
        uint64_t temp = a[i] - b[i] - borrow;
        borrow = (a[i] < b[i]) || (temp > a[i]);
        res[i] = temp;
    }
}

__device__ void u256_mul(uint64_t *res, const uint64_t *a, const uint64_t *b) {
    uint64_t temp[8] = {0};
    for (int i = 0; i < 4; i++) {
        uint64_t carry = 0;
        for (int j = 0; j < 4; j++) {
            uint64_t lo, hi;
            uint64_t a_part = a[i];
            uint64_t b_part = b[j];
            lo = __umul64hi(a_part, b_part);
            hi = a_part * b_part;

            uint64_t sum_lo = temp[i + j] + lo;
            uint64_t sum_hi = temp[i + j + 1] + hi + (sum_lo < temp[i + j]);
            temp[i + j] = sum_lo;
            temp[i + j + 1] = sum_hi;
        }
    }
    // Reduce modulo P
    for (int i = 0; i < 4; i++) {
        res[i] = temp[i];
    }
}

__device__ void u256_mod(uint64_t *res, const uint64_t *a, const uint64_t *mod) {
    uint64_t temp[4];
    for (int i = 3; i >= 0; i--) {
        if (a[i] > mod[i]) {
            u256_sub(temp, a, mod);
            for (int j = 0; j < 4; j++) res[j] = temp[j];
            return;
        }
    }
    for (int i = 0; i < 4; i++) res[i] = a[i];
}

// ECC point addition
__device__ void ec_point_add(uint64_t *rx, uint64_t *ry, const uint64_t *px, const uint64_t *py, const uint64_t *qx, const uint64_t *qy) {
    uint64_t lambda[4], temp1[4], temp2[4];

    // lambda = (qy - py) / (qx - px)
    u256_sub(temp1, qy, py);
    u256_sub(temp2, qx, px);
    u256_mod(temp2, temp2, SECP256K1_P);  // Modular inverse needed here for division
    u256_mul(lambda, temp1, temp2);
    u256_mod(lambda, lambda, SECP256K1_P);

    // rx = lambda^2 - px - qx
    u256_mul(temp1, lambda, lambda);
    u256_sub(temp1, temp1, px);
    u256_sub(rx, temp1, qx);
    u256_mod(rx, rx, SECP256K1_P);

    // ry = lambda * (px - rx) - py
    u256_sub(temp1, px, rx);
    u256_mul(temp1, lambda, temp1);
    u256_sub(ry, temp1, py);
    u256_mod(ry, ry, SECP256K1_P);
}

// ECC scalar multiplication
__device__ void ec_scalar_mul(uint64_t *rx, uint64_t *ry, const uint64_t *px, const uint64_t *py, const uint64_t *k) {
    uint64_t resx[4] = {0}, resy[4] = {0};
    uint64_t qx[4], qy[4];
    for (int i = 0; i < 4; i++) {
        qx[i] = px[i];
        qy[i] = py[i];
    }

    for (int i = 0; i < 256; i++) {
        if ((k[i / 64] >> (i % 64)) & 1) {
            if (resx[0] == 0 && resy[0] == 0 && resx[1] == 0 && resy[1] == 0 && resx[2] == 0 && resy[2] == 0 && resx[3] == 0 && resy[3] == 0) {
                for (int j = 0; j < 4; j++) {
                    resx[j] = qx[j];
                    resy[j] = qy[j];
                }
            } else {
                ec_point_add(resx, resy, resx, resy, qx, qy);
            }
        }
        ec_point_add(qx, qy, qx, qy, qx, qy);
    }
    for (int i = 0; i < 4; i++) {
        rx[i] = resx[i];
        ry[i] = resy[i];
    }
}

__global__ void kernel_ecc(uint64_t *d_rx, uint64_t *d_ry, uint64_t *d_k) {
    uint64_t px[4], py[4];
    for (int i = 0; i < 4; i++) {
        px[i] = SECP256K1_GX[i];
        py[i] = SECP256K1_GY[i];
    }
    ec_scalar_mul(d_rx, d_ry, px, py, d_k);
}

int main() {
    uint64_t h_k[4] = {0x12345678, 0x9abcdef0, 0x12345678, 0x9abcdef0};
    uint64_t h_rx[4], h_ry[4];
    uint64_t *d_k, *d_rx, *d_ry;

    hipMalloc((void**)&d_k, 4 * sizeof(uint64_t));
    hipMalloc((void**)&d_rx, 4 * sizeof(uint64_t));
    hipMalloc((void**)&d_ry, 4 * sizeof(uint64_t));

    hipMemcpy(d_k, h_k, 4 * sizeof(uint64_t), hipMemcpyHostToDevice);

    kernel_ecc<<<1, 1>>>(d_rx, d_ry, d_k);

    hipMemcpy(h_rx, d_rx, 4 * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_ry, d_ry, 4 * sizeof(uint64_t), hipMemcpyDeviceToHost);

    printf("Resulting Point:\n");
    for (int i = 0; i < 4; i++) {
        printf("rx[%d] = %016llx\n", i, h_rx[i]);
        printf("ry[%d] = %016llx\n", i, h_ry[i]);
    }

    hipFree(d_k);
    hipFree(d_rx);
    hipFree(d_ry);

    return 0;
}
