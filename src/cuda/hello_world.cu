
#include <hip/hip_runtime.h>
extern "C" __global__ void cuda_hello(int *out){
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i] = i;
}